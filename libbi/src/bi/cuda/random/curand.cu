/**
 * @file
 *
 * @author Lawrence Murray <lawrence.murray@csiro.au>
 * $Rev$
 * $Date$
 */
#include "hiprand.hpp"

#define CURAND_FUNC_DEF(name, dname, sname) \
  BOOST_TYPEOF(hiprand##sname) *bi::curand_##name<float>::func = hiprand##sname; \
  BOOST_TYPEOF(hiprand##dname) *bi::curand_##name<double>::func = hiprand##dname;

CURAND_FUNC_DEF(generate_uniform, GenerateUniformDouble, GenerateUniform)
CURAND_FUNC_DEF(generate_normal, GenerateNormalDouble, GenerateNormal)
