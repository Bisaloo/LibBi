/**
 * @file
 *
 * @author Lawrence Murray <lawrence.murray@csiro.au>
 * $Rev: 1253 $
 * $Date: 2011-02-01 12:03:44 +0800 (Tue, 01 Feb 2011) $
 */
#include "hiprand.hpp"

#define CURAND_FUNC_DEF(name, dname, sname) \
  BOOST_TYPEOF(hiprand##sname) *bi::curand_##name<float>::func = hiprand##sname; \
  BOOST_TYPEOF(hiprand##dname) *bi::curand_##name<double>::func = hiprand##dname;

CURAND_FUNC_DEF(generate_uniform, GenerateUniformDouble, GenerateUniform)
CURAND_FUNC_DEF(generate_normal, GenerateNormalDouble, GenerateNormal)
