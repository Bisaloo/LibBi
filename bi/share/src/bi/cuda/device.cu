/**
 * @file
 *
 * @author Lawrence Murray <lawrence.murray@csiro.au>
 * $Rev$
 * $Date$
 */
#include "device.hpp"

#include "cuda.hpp"

#include <vector>

int bi::chooseDevice(const int rank) {
  int dev, num;
  hipDeviceProp_t prop;
  std::vector<int> valid;

  /* build list of valid devices */
  CUDA_CHECKED_CALL(hipGetDeviceCount(&num));
  for (dev = 0; dev < num; ++dev) {
    CUDA_CHECKED_CALL(hipGetDeviceProperties(&prop, dev));
    if (prop.major >= 2) { // require compute 2.0 or later
      valid.push_back(dev);
    }
  }
  BI_ERROR_MSG(valid.size() > 0, "No devices of at least compute 1.3 available");

  /* select device */
  CUDA_CHECKED_CALL(hipSetDevice(valid[rank % valid.size()]));
  CUDA_CHECKED_CALL(hipGetDevice(&dev));

  return dev;
}

int bi::deviceIdealThreads() {
  return deviceOverloading()*deviceMultiprocessors()*deviceIdealThreadsPerBlock();
}

int bi::deviceIdealThreadsPerBlock() {
  int dev;
  hipDeviceProp_t prop;
  CUDA_CHECKED_CALL(hipGetDevice(&dev));
  CUDA_CHECKED_CALL(hipGetDeviceProperties(&prop, dev));

  if (prop.major == 1) {
    return 128;
  } else if (prop.major == 2) {
    return 256;
  } else {
    return 512;
  }
}

int bi::deviceMultiprocessors() {
  int dev;
  hipDeviceProp_t prop;
  CUDA_CHECKED_CALL(hipGetDevice(&dev));
  CUDA_CHECKED_CALL(hipGetDeviceProperties(&prop, dev));

  return prop.multiProcessorCount;
}
